#include "hip/hip_runtime.h"
#include <llis/ipc/shm_channel.h>

#include <thread>

void reader(llis::ipc::ShmChannelGpu* channel) {
    for (int i = 0; i < 10000; ++i) {
        int val;
        channel->read(&val, sizeof(val));
        if (val != i) {
            printf("Error! Expected: %d, Actual: %d\n", i, val);
            break;
        }
    }
}

__global__ void writer(llis::ipc::ShmChannelGpu channel) {
    for (int i = 0; i < 10000; ++i) {
        channel.write(i);
    }
}

int main() {
    llis::ipc::ShmChannelGpu channel(64);
    llis::ipc::ShmChannelGpu channel_gpu(&channel);

    std::thread reader_thr(reader, &channel);

    writer<<<1, 1>>>(std::move(channel_gpu));

    reader_thr.join();
    hipDeviceSynchronize();
}

