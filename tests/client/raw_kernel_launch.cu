#include "hip/hip_runtime.h"
#include <llis/ipc/shm_primitive_channel.h>
#include <llis/job/instrument.h>
#include <llis/ipc/defs.h>

#include <chrono>
#include <iostream>

__global__ void helloworld(int i, llis::JobId job_id, llis::ipc::Gpu2SchedChannel gpu2sched_channel) {
    llis::job::kernel_start(job_id, &gpu2sched_channel);
    llis::job::kernel_end(job_id, &gpu2sched_channel);
}

int main() {
    hipStream_t stream;
    hipStreamCreate(&stream);

    llis::ipc::Gpu2SchedChannel gpu2sched_channel(1024);

    for (int i = 0; i < 10; ++i) {
        auto start_time = std::chrono::steady_clock::now();

        helloworld<<<1, 1, 0, stream>>>(i, 0, gpu2sched_channel.fork());
        hipStreamSynchronize(stream);

        auto end_time = std::chrono::steady_clock::now();

        auto time_taken = end_time - start_time;
        std::cout << std::chrono::duration<double, std::micro>(time_taken).count() << std::endl;
    }
}

