#include "hip/hip_runtime.h"
#include <llis/ipc/shm_primitive_channel.h>
#include <llis/job/coroutine_job.h>
#include <llis/job/context.h>
#include <llis/job/instrument.h>

#include <cstdio>

__global__ void helloworld(int i, llis::JobId job_id, llis::ipc::Gpu2SchedChannel gpu2sched_channel
#ifdef LLIS_MEASURE_BLOCK_TIME
        , llis::ipc::Gpu2SchedChannel gpu2sched_block_time_channel
#endif
) {
#ifdef LLIS_MEASURE_BLOCK_TIME
    llis::job::BlockStartEndTime start_end_time;
    llis::job::kernel_start(job_id, &gpu2sched_channel, &start_end_time);
#else
    llis::job::kernel_start(job_id, &gpu2sched_channel);
#endif

    unsigned nsmid;
    asm("mov.u32 %0, %nsmid;" : "=r"(nsmid));
    printf("Hello world %d %d\n", i, nsmid);

#ifdef LLIS_MEASURE_BLOCK_TIME
    llis::job::kernel_end(job_id, &gpu2sched_channel, &gpu2sched_block_time_channel, &start_end_time);
#else
    llis::job::kernel_end(job_id, &gpu2sched_channel);
#endif
}

class HelloWorldCoroutineJob : public llis::job::CoroutineJob {
  public:
    size_t get_input_size() override {
        return 5;
    }

    size_t get_output_size() override {
        return 11;
    }

    size_t get_param_size() override {
        return 4;
    }

    void one_time_init() override {
        set_num_threads_per_block(1);
        set_smem_size_per_block(0);
        set_num_registers_per_thread(32);
    }

    void body(void* io_ptr) override {
        io_ptr_ = io_ptr;

        for (int i = 1; i <= 5; ++i) {
            set_num_blocks(i);

            yield();
            helloworld<<<i, 1, 0, get_cuda_stream()>>>(i, get_id(), llis::job::Context::get_gpu2sched_channel()->fork()
#ifdef LLIS_MEASURE_BLOCK_TIME
                , llis::job::Context::get_gpu2sched_block_time_channel()->fork()
#endif
                );
        }
    }

  private:
    void* io_ptr_;
};

extern "C" {

llis::job::Job* init_job() {
    return new HelloWorldCoroutineJob();
}

}

