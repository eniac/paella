#include "hip/hip_runtime.h"
#include <llis/job/coroutine_job.h>
#include <llis/job/context.h>
#include <llis/job/instrument.h>

#include <cstdio>

__global__ void helloworld(int i, void* job, llis::ipc::ShmChannelGpu gpu2sched_channel) {
    llis::job::kernel_start(job, &gpu2sched_channel);

    unsigned nsmid;
    asm("mov.u32 %0, %nsmid;" : "=r"(nsmid));
    printf("Hello world %d %d\n", i, nsmid);

    llis::job::kernel_end(job, &gpu2sched_channel);
}

class HelloWorldCoroutineJob : public llis::job::CoroutineJob {
  public:
    HelloWorldCoroutineJob() {
        set_num_threads_per_block(1);
        set_smem_size_per_block(0);
        set_num_registers_per_thread(32);
    }

    size_t get_input_size() override {
        return 5;
    }

    size_t get_output_size() override {
        return 11;
    }

    size_t get_param_size() override {
        return 4;
    }

    void full_init(void* io_ptr) override {
        CoroutineJob::full_init(io_ptr);

        io_ptr_ = io_ptr;
    }

    void body() override {
        for (int i = 0; i < 5; ++i) {
            ++num_;
            set_num_blocks(num_);

            yield();
            num_running_blocks_ = num_;
            helloworld<<<num_running_blocks_, 1, 0, get_cuda_stream()>>>(num_, this, llis::job::Context::get_gpu2sched_channel()->fork());
        }
    }

    void mark_block_finish() override {
        num_running_blocks_--;
        if (num_running_blocks_ == 0) {
            unset_running();
        }
    }

  private:
    void* io_ptr_;
    int num_ = 0;
    int num_running_blocks_;
};

extern "C" {

llis::job::Job* init_job() {
    return new HelloWorldCoroutineJob();
}

}

