#include "hip/hip_runtime.h"
#include <llis/ipc/shm_primitive_channel.h>
#include <llis/job/coroutine_job.h>
#include <llis/job/context.h>
#include <llis/job/instrument.h>

#include <cstdio>

__global__ void vec_add(float* output, float* input, size_t count, unsigned long long dummy[10], llis::JobId job_id, llis::ipc::Gpu2SchedChannel gpu2sched_channel
#ifdef LLIS_MEASURE_BLOCK_TIME
        , llis::ipc::Gpu2SchedChannel gpu2sched_block_time_channel
#endif
) {
#ifdef LLIS_MEASURE_BLOCK_TIME
    llis::job::BlockStartEndTime start_end_time;
    llis::job::kernel_start(job_id, &gpu2sched_channel, &start_end_time);
#else
    llis::job::kernel_start(job_id, &gpu2sched_channel);
#endif

    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned grid_size = blockDim.x * gridDim.x;

    while (id < count) {
        output[id] += input[id];
        id += grid_size;
    }

#ifdef LLIS_MEASURE_BLOCK_TIME
    llis::job::kernel_end(job_id, &gpu2sched_channel, &gpu2sched_block_time_channel, &start_end_time);
#else
    llis::job::kernel_end(job_id, &gpu2sched_channel);
#endif
}

class VecAddCoroutineJob : public llis::job::CoroutineJob {
  private:
    //static constexpr unsigned count_ = 128000;
    static constexpr unsigned count_ = 32000;
    static constexpr unsigned num_blocks = 200;
    //static constexpr unsigned num_blocks = 1000;
    //static constexpr unsigned num_threads_per_block = count_ / num_blocks;
    static constexpr unsigned num_threads_per_block = 10;
    //static constexpr unsigned num_iters = 55;
    static constexpr unsigned num_iters = 10;

    static constexpr unsigned num_blocks_list[] = {196,
        224,
        98,
        392,
        2688,
        28,
        168,
        504,
        28,
        168,
        2016,
        4,
        168,
        192,
        7,
        168,
        192,
        7,
        168,
        224,
        4,
        32,
        384,
        7,
        32,
        384,
        7,
        32,
        384,
        7,
        32,
        384,
        7,
        112,
        1008,
        12,
        112,
        1008,
        12,
        112,
        168,
        8,
        12,
        48,
        8,
        12,
        48,
        8,
        12,
        48,
        5,
        20,
        160,
        2,
        1000};

  public:
    size_t get_input_size() override {
        return count_ * sizeof(float);
    }

    size_t get_output_size() override {
        return count_ * sizeof(float);
    }

    size_t get_param_size() override {
        return 4;
    }

    void one_time_init() override {
        set_num_threads_per_block(num_threads_per_block);
        set_num_blocks(num_blocks);
        set_smem_size_per_block(0);

        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(vec_add));
        set_num_registers_per_thread(attr.numRegs);

        hipMalloc(&input_dev_, get_input_size());
        hipMalloc(&output_dev_, get_output_size());
    }

    void body(void* io_ptr) override {
        float* input = (float*)io_ptr;
        float* output = (float*)io_ptr + count_;

        set_is_mem();
        yield();
        hipMemcpyAsync(input_dev_, input, get_input_size(), hipMemcpyHostToDevice, get_cuda_stream());
        unset_is_mem();

        unsigned long long dummy[10];

        //set_pre_notify();

        for (int i = 0; i < num_iters; ++i) {
            //unsigned num_blocks = num_blocks_list[i];
            //unsigned num_threads_per_block = 100;
            //set_num_blocks(num_blocks);
            //set_num_threads_per_block(num_threads_per_block);
            yield();
            vec_add<<<num_blocks, num_threads_per_block, 0, get_cuda_stream()>>>(output, input, count_, dummy, get_id(), llis::job::Context::get_gpu2sched_channel()->fork()
#ifdef LLIS_MEASURE_BLOCK_TIME
                , llis::job::Context::get_gpu2sched_block_time_channel()->fork()
#endif
                );
        }

        set_is_mem();
        set_pre_notify();
        yield();
        hipMemcpyAsync(output, output_dev_, get_output_size(), hipMemcpyDeviceToHost, get_cuda_stream());
    }

  private:
    float* input_dev_;
    float* output_dev_;
};

extern "C" {

llis::job::Job* init_job() {
    return new VecAddCoroutineJob();
}

}

