#include "hip/hip_runtime.h"
#include <llis/ipc/shm_primitive_channel.h>
#include <llis/job/coroutine_job.h>
#include <llis/job/context.h>
#include <llis/job/instrument.h>

#include <cstdio>

__global__ void dummy_kernel(float* mem, unsigned count, unsigned compute_count, llis::JobId job_id, llis::job::FinishedBlockNotifier* notifier) {
    notifier->start(job_id);

    //clock_t start_time = clock64();
    //while (clock64() - start_time < 10000000);

    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned grid_size = blockDim.x * gridDim.x;

    while (id < count) {
        float tmp = 1;
        for (unsigned i = 1; i <= compute_count; ++i) {
            tmp *= i;
        }
        mem[id] = tmp;
        id += grid_size;
    }

    notifier->end(job_id);
}

class DummyShortCoroutineJob : public llis::job::CoroutineJob {
  public:
    size_t get_input_size() override {
        return 5;
    }

    size_t get_output_size() override {
        return 11;
    }

    size_t get_param_size() override {
        return 4;
    }

    void one_time_init() override {
        set_num_threads_per_block(256);
        set_smem_size_per_block(0);
        set_num_registers_per_thread(32);
        set_num_blocks(5);
        unset_is_mem();

        hipMalloc(&mem_, count_ * sizeof(*mem_));
    }

    void body(void* io_ptr) override {
        for (int i = 0; i < num_kernels; ++i) {
            if (i == num_kernels - 1) {
                set_pre_notify();
            }
            yield();
            llis::job::FinishedBlockNotifier* notifier = get_finished_block_notifier();
            dummy_kernel<<<get_num_blocks(), get_num_threads_per_block(), 0, get_cuda_stream()>>>(mem_, count_, compute_count_, get_id(), notifier);
        }
    }

  private:
    float* mem_;

    static constexpr unsigned count_ = 5000000;
    static constexpr unsigned compute_count_ = 100;
    static constexpr unsigned num_kernels = 11;
};

extern "C" {

__attribute__((visibility("default")))
llis::job::Job* init_job() {
    return new DummyShortCoroutineJob();
}

}

